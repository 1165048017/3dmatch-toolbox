#define DATATYPE 0
#include "marvin.hpp"
#include <opencv2/opencv.hpp>

int main(int argc, char * argv[]) {

	std::cout << argv[1] << std::endl;
	std::string filename(argv[1]);

	// Read binary file containing TDF voxel grid values
    FILE * fp = fopen(argv[1],"rb");
    float numVolumesf;
    float volumeGridSizeXf;
    float volumeGridSizeYf;
    float volumeGridSizeZf;
	int iret = fread((void*)(&numVolumesf), sizeof(float), 1, fp);
	int numVolumes = (int)numVolumesf;
	iret = fread((void*)(&volumeGridSizeXf), sizeof(float), 1, fp);
	iret = fread((void*)(&volumeGridSizeYf), sizeof(float), 1, fp);
	iret = fread((void*)(&volumeGridSizeZf), sizeof(float), 1, fp);
    int numVolumeGridPoints = (int)(volumeGridSizeXf * volumeGridSizeYf * volumeGridSizeZf);
    float * tudfVoxf = new float[numVolumes*numVolumeGridPoints];
	iret = fread((void*)(tudfVoxf), sizeof(float), numVolumes*numVolumeGridPoints, fp);
	fclose(fp);

    // Start Marvin network
	marvin::Net convnet("tmp.json");
	convnet.Malloc(marvin::Testing);
	convnet.loadWeights("3dmatch-weights-snapshot-127000-apc-16000.marvin");
	marvin::Response * rData;
	marvin::Response * rFeat;
	rData = convnet.getResponse("data");
	rFeat = convnet.getResponse("feat");

	fp = fopen("feat.bin", "wb");
	fwrite(&numVolumesf, sizeof(float), 1, fp);
	for (int i = 1; i < 5; i++) {
		float value = (float)(rFeat->dim[i]);
		fwrite(&value, sizeof(float), 1, fp);
	}

	std::cout << numVolumes << std::endl;
    StorageT * itudfVox = new StorageT[50*numVolumeGridPoints];
	int numFeatVolumeGridPoints = rFeat->dim[0] * rFeat->dim[1] * rFeat->dim[2] * rFeat->dim[3] * rFeat->dim[4];
	StorageT * featVolume = new StorageT[numFeatVolumeGridPoints];

	for (int vox_idx = 0; vox_idx < numVolumes/50; ++vox_idx) {
	    for (int i = 0; i < 50*numVolumeGridPoints; ++i) {
	    	itudfVox[i] = CPUCompute2StorageT(tudfVoxf[vox_idx*50*numVolumeGridPoints + i]);
		}
	    
		hipMemcpy(rData->dataGPU, itudfVox, rData->numBytes(), hipMemcpyHostToDevice);
		marvin::checkCUDA(__LINE__, hipGetLastError());

		convnet.forward();

		hipMemcpy(featVolume, rFeat->dataGPU, rFeat->numBytes(), hipMemcpyDeviceToHost);

		for (int i = 0; i < numFeatVolumeGridPoints; i++) {
			float value = CPUStorage2ComputeT(featVolume[i]);
			fwrite(&value, sizeof(float), 1, fp);
		}

	}
	fclose(fp);

	return 0;
}
















































